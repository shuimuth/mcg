#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <map>
#include <iostream>
#include <set>
#include <utility>
#include <vector>

#include <hip/hip_runtime.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>      // helper function CUDA error checking and initialization
#include <helper_functions.h> // helper for shared functions common to CUDA Samples

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

#define ENABLE_CPU_DEBUG_CODE 0
#define THREADS_PER_BLOCK 512

__device__ double grid_dot_result = 0.0;

__device__ double atomicAdd_(double *address, double val)
{
  unsigned long long int *address_as_ull =
      (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;

  do
  {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val +
                                         __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz)
{
  I[0] = 0, J[0] = 0, J[1] = 1;
  val[0] = (float)rand() / RAND_MAX + 10.0f;
  val[1] = (float)rand() / RAND_MAX;
  int start;

  for (int i = 1; i < N; i++)
  {
    if (i > 1)
    {
      I[i] = I[i - 1] + 3;
    }
    else
    {
      I[1] = 2;
    }

    start = (i - 1) * 3 + 2;
    J[start] = i - 1;
    J[start + 1] = i;

    if (i < N - 1)
    {
      J[start + 2] = i + 1;
    }

    val[start] = val[start - 1];
    val[start + 1] = (float)rand() / RAND_MAX + 10.0f;

    if (i < N - 1)
    {
      val[start + 2] = (float)rand() / RAND_MAX;
    }
  }

  I[N] = nz;
}

// I - contains location of the given non-zero element in the row of the matrix
// J - contains location of the given non-zero element in the column of the
// matrix val - contains values of the given non-zero elements of the matrix
// inputVecX - input vector to be multiplied
// outputVecY - resultant vector
void cpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha,
             float *inputVecX, float *outputVecY)
{
  for (int i = 0; i < num_rows; i++)
  {
    int num_elems_this_row = I[i + 1] - I[i];

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++)
    {
      output += alpha * val[I[i] + j] * inputVecX[J[I[i] + j]];
    }
    outputVecY[i] = output;
  }

  return;
}

float dotProduct(float *vecA, float *vecB, int size)
{
  float result = 0.0;

  for (int i = 0; i < size; i++)
  {
    result = result + (vecA[i] * vecB[i]);
  }

  return result;
}

void scaleVector(float *vec, float alpha, int size)
{
  for (int i = 0; i < size; i++)
  {
    vec[i] = alpha * vec[i];
  }
}

void saxpy(float *x, float *y, float a, int size)
{
  for (int i = 0; i < size; i++)
  {
    y[i] = a * x[i] + y[i];
  }
}

void cpuConjugateGrad(int *I, int *J, float *val, float *x, float *Ax, float *p,
                      float *r, int nnz, int N, float tol)
{
  int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, b, a, na;

  cpuSpMV(I, J, val, nnz, N, alpha, x, Ax);
  saxpy(Ax, r, alpham1, N);

  float r1 = dotProduct(r, r, N);

  int k = 1;

  while (r1 > tol * tol && k <= max_iter)
  {
    if (k > 1)
    {
      b = r1 / r0;
      scaleVector(p, b, N);

      saxpy(r, p, alpha, N);
    }
    else
    {
      for (int i = 0; i < N; i++)
        p[i] = r[i];
    }

    cpuSpMV(I, J, val, nnz, N, alpha, p, Ax);

    float dot = dotProduct(p, Ax, N);
    a = r1 / dot;

    saxpy(p, x, a, N);
    na = -a;
    saxpy(Ax, r, na, N);

    r0 = r1;
    r1 = dotProduct(r, r, N);

    printf("\nCPU code iteration = %3d, residual = %e\n", k, sqrt(r1));
    k++;
  }
}

__global__ void gpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, int startIdx,
                        float *inputVecX, float *outputVecY)
{
  cg::grid_group grid = cg::this_grid();

  for (int i = grid.thread_rank(); i < num_rows; i += grid.size())
  {
    int globalIdx = startIdx + i;
    int row_elem = I[globalIdx];
    int next_row_elem = I[globalIdx + 1];
    int num_elems_this_row = next_row_elem - row_elem;

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++)
    {
      output += val[row_elem + j] * inputVecX[J[row_elem + j]];
    }

    outputVecY[globalIdx] = output;
    // printf("%d Ax[%d] = %f\n", globalIdx, globalIdx, output);
  }
}

__global__ void gpuSaxpy(float *x, float *y, float a, int size, int startIdx)
{

  cg::grid_group grid = cg::this_grid();
  for (int i = grid.thread_rank(); i < size; i += grid.size())
  {
    int globalIdx = startIdx + i;
    y[globalIdx] = a * x[globalIdx] + y[globalIdx];
  }
}

__device__ void gpuScaleVectorAndSaxpy(float *x, float *y, float a, float scale, int size, int startIdx)
{
  cg::grid_group grid = cg::this_grid();
  for (int i = grid.thread_rank(); i < size; i += grid.size())
  {
    int globalIdx = startIdx + i;
    y[i] = a * x[i] + scale * y[i];
  }
}

// __global__ void gpuDotProduct(double* result, float* vecA, float* vecB, int size, int startIdx) {
//     cg::grid_group grid = cg::this_grid();

//     for (int i = grid.thread_rank(); i < size; i += grid.size()) {
//         int globalIdx = startIdx + i;
//         double tmp = static_cast<double>(vecA[globalIdx] * vecB[globalIdx]);
//         atomicAdd_(result, tmp);
//     }
// }

__global__ void gpuDotProduct(double *result, float *vecA, float *vecB, int size, int startIdx)
{
  __shared__ double tmp[THREADS_PER_BLOCK];

  cg::thread_block cta = cg::this_thread_block();
  cg::grid_group grid = cg::this_grid();

  double temp_sum = 0.0;
  for (int i = grid.thread_rank(); i < size; i += grid.size())
  {
    int globalIdx = startIdx + i;
    temp_sum += static_cast<double>(vecA[globalIdx] * vecB[globalIdx]);
  }
  tmp[cta.thread_rank()] = temp_sum;

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  double beta = temp_sum;
  double temp;

  for (int i = tile32.size() / 2; i > 0; i >>= 1)
  {
    if (tile32.thread_rank() < i)
    {
      temp = tmp[cta.thread_rank() + i];
      beta += temp;
      tmp[cta.thread_rank()] = beta;
    }
    cg::sync(tile32);
  }
  cg::sync(cta);

  if (cta.thread_rank() == 0)
  {
    beta = 0.0;
    for (int i = 0; i < cta.size(); i += tile32.size())
    {
      beta += tmp[i];
    }
    atomicAdd_(result, beta);
  }
}

class SPMat
{
public:
  int *I;
  int *J;
  int N;
  int nz;
  float *val;

public:
  void init(int size, int noneZero)
  {
    N = size;
    nz = noneZero;
    mallocGpuMem();
  }

  void mallocGpuMem()
  {
    hipMalloc(&I, (N + 1) * sizeof(int));
    hipMalloc(&J, nz * sizeof(int));
    hipMalloc(&val, nz * sizeof(float));
  }

  void destroy()
  {
    hipFree(I);
    hipFree(J);
    hipFree(val);
  }
};

class CGData
{
public:
  float *r;
  float *p;
  float *Ax;
  float *x;
  double *dotProduct;
  int size;
  int localNumRow;
  int deviceId;
  int global_begin_idx;
  hipStream_t stream;

  SPMat spMat;

public:
  void init(int id, int size_, int nnz, int gbi, int numRow)
  {
    deviceId = id;
    size = size_;
    global_begin_idx = gbi;
    localNumRow = numRow;
    spMat.init(size_, nnz);
    mallocGpuMem();
  }

  // void setDeviceId(int id) { deviceId = id; }
  // void setSize(int size_) { size = size_; }
  // void setGlobalBeginIdx(int idx) { global_begin_idx = idx; }

  void mallocGpuMem()
  {
    checkCudaErrors(hipSetDevice(deviceId));
    hipMalloc(&r, size * sizeof(float));
    hipMalloc(&p, size * sizeof(float));
    hipMalloc(&Ax, size * sizeof(float));
    hipMalloc(&x, size * sizeof(float));
    hipMalloc(&dotProduct, sizeof(double));
    spMat.mallocGpuMem();
  }

  void destroy()
  {
    hipFree(r);
    hipFree(p);
    hipFree(Ax);
    hipFree(x);
    hipFree(dotProduct);
    spMat.destroy();
  }
};

class MultiCGData
{
public:
  int numDevice;
  int numRow;
  CGData *cgData;
  hipStream_t *stream;

  MultiCGData(const int nd, const int N, const int nnz)
  {
    numDevice = nd;
    numRow = N;
    cgData = new CGData[numDevice];

    int rowPerDevice = N / nd;
    int rowLastDevice = N - rowPerDevice * (nd - 1);

    for (int i = 0; i < numDevice; i++)
    {
      int rowDevice = i == numDevice - 1 ? rowLastDevice : rowPerDevice;
      cgData[i].init(i, N, nnz, i * rowPerDevice, rowDevice);
    }

    stream = new hipStream_t[numDevice];
  }

  void createCudaStream()
  {
    for (int i = 0; i < numDevice; i++)
    {
      hipSetDevice(i);
      checkCudaErrors(hipStreamCreate(&stream[i]));
    }
  }

  void destroyCudaStream()
  {
    for(int i = 0; i < numDevice; i++)
    {
      checkCudaErrors(hipStreamDestroy(stream[i]));
    }
  }

  ~MultiCGData()
  {
    for (int i = 0; i < numDevice; i++)
    {
      cgData[i].destroy();
    }
  }
};

void multiGpuSpMV(MultiCGData &multiCGData)
{
  dim3 dimGrid(32, 1, 1);
  dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);

  for (int i = 0; i < multiCGData.numDevice; i++)
  {
    CGData &data = multiCGData.cgData[i];

    checkCudaErrors(hipSetDevice(data.deviceId));

    void *kernelArgs[] = {
        (void *)&data.spMat.I,
        (void *)&data.spMat.J,
        (void *)&data.spMat.val,
        (void *)&data.spMat.nz,
        (void *)&data.localNumRow,
        (void *)&data.global_begin_idx,
        (void *)&data.x,
        (void *)&data.Ax};

    std::cout << "device " << data.deviceId << std::endl;

    checkCudaErrors(hipLaunchCooperativeKernel(
        (void *)gpuSpMV, dimGrid, dimBlock, kernelArgs,
        0, multiCGData.stream[data.deviceId]));
    getLastCudaError("gpuSpMV execution failed");
  }
}

void multiGpuSaxpy(MultiCGData &multiCGData)
{
  dim3 dimGrid(32, 1, 1);
  dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);

  for (int i = 0; i < multiCGData.numDevice; i++)
  {
    CGData &data = multiCGData.cgData[i];

    checkCudaErrors(hipSetDevice(data.deviceId));

    float alpha = -1.0;
    void *sapyArgs[] = {
        &data.Ax,
        &data.r,
        &alpha,
        &data.localNumRow,
        &data.global_begin_idx};

    checkCudaErrors(hipLaunchCooperativeKernel(
        (void *)gpuSaxpy, dimGrid, dimBlock, sapyArgs,
        0, multiCGData.stream[data.deviceId]));
    getLastCudaError("gpuSaxpy execution failed");
  }
}

double multiGpuDotProductRR(MultiCGData& multiCGData)
{
  dim3 dimGrid(32, 1, 1);
  dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);

  std::vector<double> dotResult(multiCGData.numDevice);
  for (int i = 0; i < multiCGData.numDevice; i++)
  {
    CGData &data = multiCGData.cgData[i];

    checkCudaErrors(hipSetDevice(data.deviceId));

    void *dotProductArgs[] = {
        &data.dotProduct,
        &data.r,
        &data.r,
        &data.localNumRow,
        &data.global_begin_idx};

    int sMemSize = sizeof(double) * THREADS_PER_BLOCK;
    checkCudaErrors(hipLaunchCooperativeKernel(
        (void *)gpuDotProduct, dimGrid, dimBlock, dotProductArgs,
        0, multiCGData.stream[data.deviceId]));
    getLastCudaError("gpuDotProduct execution failed");

    checkCudaErrors(hipMemcpyAsync(dotResult.data() + data.deviceId, data.dotProduct, sizeof(double), hipMemcpyDefault, multiCGData.stream[data.deviceId]));
  }

  double sum = 0.0;
  for(int i = 0; i < multiCGData.numDevice; i++)
  {
    checkCudaErrors(hipSetDevice(i));
    hipDeviceSynchronize();
    sum += dotResult[i];
  }
  return sum;
}

__global__ void setValue(float* x, float val, int count)
{
  cg::grid_group grid = cg::this_grid();
  for(int i = grid.thread_rank(); i < count; i += grid.size())
  {
    x[i] = val;
  }
}

void initGpuData(MultiCGData& multiCGData)
{
  dim3 dimGrid(32, 1, 1);
  dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);
  for(int i = 0; i < multiCGData.numDevice; i++)
  {
    CGData &data = multiCGData.cgData[i];

    checkCudaErrors(hipSetDevice(data.deviceId));
    cudaMemSet(data.x, 0, multiCGData.numRow * sizeof(float));

    float val = 1.0;
    void* args[] = {&data.r, &val, &multiCGData.numRow};
    checkCudaErrors(hipLaunchCooperativeKernel(
                    (void *)setValue, dimGrid, dimBlock, args,
                    0, multiCGData.stream[data.deviceId]));
    getLastCudaError("gpuSaxpy execution failed");
  }
}

void ConjugateGrad(int *I, int *J, float *val, float *x, float *Ax, float *p,
                   float *r, int nnz, int N, float tol, MultiCGData& multiCGData)
{
  int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, b, a, na;



  cpuSpMV(I, J, val, nnz, N, alpha, x, Ax);
  multiGpuSpMV(multiCGData);
  
  saxpy(Ax, r, alpham1, N);
  multiGpuSaxpy(multiCGData);

  float r1 = dotProduct(r, r, N);

  auto r1_ = multiGpuDotProductRR(multiCGData);


  int k = 1;

  while (r1 > tol * tol && k <= max_iter)
  {
    if (k > 1)
    {
      b = r1 / r0;
      scaleVector(p, b, N);

      saxpy(r, p, alpha, N);
    }
    else
    {
      for (int i = 0; i < N; i++)
        p[i] = r[i];
    }

    cpuSpMV(I, J, val, nnz, N, alpha, p, Ax);

    float dot = dotProduct(p, Ax, N);
    a = r1 / dot;

    saxpy(p, x, a, N);
    na = -a;
    saxpy(Ax, r, na, N);

    r0 = r1;
    r1 = dotProduct(r, r, N);

    printf("\nCPU code iteration = %3d, residual = %e\n", k, sqrt(r1));
    k++;
  }
}

// Map of device version to device number
std::multimap<std::pair<int, int>, int> getIdenticalGPUs()
{
  int numGpus = 0;
  checkCudaErrors(hipGetDeviceCount(&numGpus));

  std::multimap<std::pair<int, int>, int> identicalGpus;

  for (int i = 0; i < numGpus; i++)
  {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, i));

    // Filter unsupported devices
    if (deviceProp.cooperativeLaunch)
    {
      identicalGpus.emplace(std::make_pair(deviceProp.major, deviceProp.minor),
                            i);
    }
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", i,
           deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  return identicalGpus;
}

void initDevice(const int kNumGpusRequired)
{
  auto gpusByArch = getIdenticalGPUs();

  auto it = gpusByArch.begin();
  auto end = gpusByArch.end();

  auto bestFit = std::make_pair(it, it);
  // use std::distance to find the largest number of GPUs amongst architectures
  auto distance = [](decltype(bestFit) p)
  {
    return std::distance(p.first, p.second);
  };

  // Read each unique key/pair element in order
  for (; it != end; it = gpusByArch.upper_bound(it->first))
  {
    // first and second are iterators bounded within the architecture group
    auto testFit = gpusByArch.equal_range(it->first);
    // Always use devices with highest architecture version or whichever has the
    // most devices available
    if (distance(bestFit) <= distance(testFit))
      bestFit = testFit;
  }

  if (distance(bestFit) < kNumGpusRequired)
  {
    printf(
        "No two or more GPUs with same architecture capable of "
        "concurrentManagedAccess found. "
        "\nWaiving the sample\n");
    exit(EXIT_WAIVED);
  }

  std::set<int> bestFitDeviceIds;

  // Check & select peer-to-peer access capable GPU devices as enabling p2p
  // access between participating GPUs gives better performance.
  for (auto itr = bestFit.first; itr != bestFit.second; itr++)
  {
    int deviceId = itr->second;
    checkCudaErrors(hipSetDevice(deviceId));

    std::for_each(
        itr, bestFit.second,
        [&deviceId, &bestFitDeviceIds,
         &kNumGpusRequired](decltype(*itr) mapPair)
        {
          if (deviceId != mapPair.second)
          {
            int access = 0;
            checkCudaErrors(
                hipDeviceCanAccessPeer(&access, deviceId, mapPair.second));
            printf("Device=%d %s Access Peer Device=%d\n", deviceId,
                   access ? "CAN" : "CANNOT", mapPair.second);
            if (access && bestFitDeviceIds.size() < kNumGpusRequired)
            {
              bestFitDeviceIds.emplace(deviceId);
              bestFitDeviceIds.emplace(mapPair.second);
            }
            else
            {
              printf("Ignoring device %i (max devices exceeded)\n",
                     mapPair.second);
            }
          }
        });

    if (bestFitDeviceIds.size() >= kNumGpusRequired)
    {
      printf("Selected p2p capable devices - ");
      for (auto devicesItr = bestFitDeviceIds.begin();
           devicesItr != bestFitDeviceIds.end(); devicesItr++)
      {
        printf("deviceId = %d  ", *devicesItr);
      }
      printf("\n");
      break;
    }
  }

  // if bestFitDeviceIds.size() == 0 it means the GPUs in system are not p2p
  // capable, hence we add it without p2p capability check.
  if (!bestFitDeviceIds.size())
  {
    printf("Devices involved are not p2p capable.. selecting %zu of them\n",
           kNumGpusRequired);
    std::for_each(bestFit.first, bestFit.second,
                  [&bestFitDeviceIds,
                   &kNumGpusRequired](decltype(*bestFit.first) mapPair)
                  {
                    if (bestFitDeviceIds.size() < kNumGpusRequired)
                    {
                      bestFitDeviceIds.emplace(mapPair.second);
                    }
                    else
                    {
                      printf("Ignoring device %i (max devices exceeded)\n",
                             mapPair.second);
                    }
                    // Insert the sequence into the deviceIds set
                  });
  }
  else
  {
    // perform hipDeviceEnablePeerAccess in both directions for all
    // participating devices.
    for (auto p1_itr = bestFitDeviceIds.begin();
         p1_itr != bestFitDeviceIds.end(); p1_itr++)
    {
      checkCudaErrors(hipSetDevice(*p1_itr));
      for (auto p2_itr = bestFitDeviceIds.begin();
           p2_itr != bestFitDeviceIds.end(); p2_itr++)
      {
        if (*p1_itr != *p2_itr)
        {
          checkCudaErrors(hipDeviceEnablePeerAccess(*p2_itr, 0));
          checkCudaErrors(hipSetDevice(*p1_itr));
        }
      }
    }
  }
}

int main(int, char **)
{
  constexpr size_t kNumGpusRequired = 4;
  initDevice(kNumGpusRequired);

  int N = 0, nz = 0, *I = NULL, *J = NULL;
  float *val = NULL;
  const float tol = 1e-5f;
  float *x;
  float rhs = 1.0;
  float r1;
  float *r, *p, *Ax;

  /* Generate a random tridiagonal symmetric matrix in CSR format */
  N = 128;
  nz = (N - 2) * 3 + 4;

  I = new int[N + 1];
  J = new int[nz];
  val = new float[nz];

  genTridiag(I, J, val, N, nz);

  ////////////////////////////////////////////////////////

  std::vector<float> invec(N, 1.0);
  std::vector<float> out(N);
  std::vector<double> dotResult(kNumGpusRequired);

  MultiCGData multiCGData(kNumGpusRequired, N, nz);
  for (int i = 0; i < multiCGData.numDevice; i++)
  {
    hipSetDevice(multiCGData.cgData[i].deviceId);
    checkCudaErrors(hipMemcpy(multiCGData.cgData[i].spMat.I, I, sizeof(int) * (N + 1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(multiCGData.cgData[i].spMat.J, J, sizeof(int) * nz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(multiCGData.cgData[i].spMat.val, val, sizeof(float) * nz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(multiCGData.cgData[i].x, invec.data(), sizeof(float) * N, hipMemcpyHostToDevice));
  }

  cpuSpMV(I, J, val, nz, N, 1, invec.data(), out.data());
  saxpy(invec.data(), out.data(), 1, out.size());
  float dotRes = dotProduct(invec.data(), out.data(), out.size());

  hipStream_t nStreams[kNumGpusRequired];
  for (int i = 0; i < multiCGData.numDevice; i++)
  {
    hipSetDevice(i);
    checkCudaErrors(hipStreamCreate(&nStreams[i]));
  }

  dim3 dimGrid(32, 1, 1);
  dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);

  printf("Launching kernel\n");

  for (int i = 0; i < multiCGData.numDevice; i++)
  {
    CGData &data = multiCGData.cgData[i];

    checkCudaErrors(hipSetDevice(data.deviceId));

    void *kernelArgs[] = {
        (void *)&data.spMat.I,
        (void *)&data.spMat.J,
        (void *)&data.spMat.val,
        (void *)&data.spMat.nz,
        (void *)&data.localNumRow,
        (void *)&data.global_begin_idx,
        (void *)&data.x,
        (void *)&data.Ax};

    std::cout << "device " << data.deviceId << std::endl;

    checkCudaErrors(hipLaunchCooperativeKernel(
        (void *)gpuSpMV, dimGrid, dimBlock, kernelArgs,
        0, nStreams[data.deviceId]));
    getLastCudaError("gpuSpMV execution failed");

    float alpha = 1.0;
    void *sapyArgs[] = {
        &data.x,
        &data.Ax,
        &alpha,
        &data.localNumRow,
        &data.global_begin_idx};

    checkCudaErrors(hipLaunchCooperativeKernel(
        (void *)gpuSaxpy, dimGrid, dimBlock, sapyArgs,
        0, nStreams[data.deviceId]));
    getLastCudaError("gpuSaxpy execution failed");

    void *dotProductArgs[] = {
        &data.r,
        &data.Ax,
        &data.x,
        &data.localNumRow,
        &data.global_begin_idx};

    int sMemSize = sizeof(double) * THREADS_PER_BLOCK;
    checkCudaErrors(hipLaunchCooperativeKernel(
        (void *)gpuDotProduct, dimGrid, dimBlock, dotProductArgs,
        0, nStreams[data.deviceId]));
    getLastCudaError("gpuDotProduct execution failed");

    // transfer data to other device
    for (int j = 1; j < multiCGData.numDevice; j++)
    {
      auto &nextData = multiCGData.cgData[(i + j) % multiCGData.numDevice];
      checkCudaErrors(hipMemcpyAsync(nextData.Ax + data.global_begin_idx, data.Ax + data.global_begin_idx,
                                      data.localNumRow * sizeof(float), hipMemcpyDefault, nStreams[data.deviceId]));
    }

    checkCudaErrors(hipMemcpyAsync(dotResult.data() + data.deviceId, data.r, sizeof(double), hipMemcpyDefault, nStreams[data.deviceId]));
  }

  std::cout << "I" << std::endl;
  for (int i = 0; i < N + 1; i++)
  {
    std::cout << I[i] << " ";
  }
  std::cout << std::endl;

  std::cout << "j" << std::endl;
  for (int i = 0; i < nz; i++)
  {
    std::cout << J[i] << " ";
  }
  std::cout << std::endl;

  std::cout << "val" << std::endl;
  for (int i = 0; i < nz; i++)
  {
    std::cout << val[i] << " ";
  }
  std::cout << std::endl;

  std::cout << "MV output" << std::endl;
  for (int i = 0; i < N; i++)
  {
    std::cout << out[i] << " ";
  }
  std::cout << std::endl;

  for (int i = 0; i < kNumGpusRequired; i++)
  {
    checkCudaErrors(hipStreamSynchronize(nStreams[i]));
  }

  std::vector<float> Ax_cpu(N);
  for (int i = 0; i < multiCGData.numDevice; i++)
  {
    auto &data = multiCGData.cgData[i];
    checkCudaErrors(hipMemcpy(Ax_cpu.data(), data.Ax, Ax_cpu.size() * sizeof(float), hipMemcpyDefault));

    std::cout << "device " << i << std::endl;
    for (int idx = 0; idx < Ax_cpu.size(); idx++)
    {
      std::cout << Ax_cpu[idx] << " ";
    }
    std::cout << std::endl;
  }

  std::cout << "------------dot result--------------" << std::endl;
  std::cout << "cpu dotproduct: " << dotRes << std::endl;

  double gpuDotSum = 0.0;
  for (int i = 0; i < dotResult.size(); i++)
  {
    gpuDotSum += dotResult[i];
    std::cout << dotResult[i] << " ";
  }
  std::cout << std::endl;
  std::cout << "gpu dotproduct: " << gpuDotSum << std::endl;
  return 0;
}
